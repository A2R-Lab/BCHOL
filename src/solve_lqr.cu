#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <numeric>
#include <cmath>
#include "solve.cuh"
#include "helpf.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <vector>
#include "gpu_assert.cuh"


__host__ int main()
{
  printf("Run Test\n");
  // Declaration of LQR problem
  uint32_t knot_points = 8;
  uint32_t state_size = 6;
  uint32_t control_size = 3;
  uint32_t depth = log2(knot_points);

  // calculating the constants
  const uint32_t states_sq = state_size * state_size;
  const uint32_t controls_sq = control_size * control_size;
  const uint32_t states_p_controls = state_size * control_size;
  const uint32_t states_s_controls = state_size + control_size;
  const uint32_t fstates_size = states_sq * knot_points * depth;
  const uint32_t fcontrol_size = states_p_controls * knot_points * depth;

  const uint32_t KKT_G_DENSE_SIZE_BYTES = static_cast<uint32_t>(((states_sq + controls_sq) * knot_points - controls_sq) * sizeof(float));
  const uint32_t KKT_C_DENSE_SIZE_BYTES = static_cast<uint32_t>((states_sq + states_p_controls) * (knot_points - 1) * sizeof(float));
  const uint32_t KKT_g_SIZE_BYTES = static_cast<uint32_t>(((state_size + control_size) * knot_points - control_size) * sizeof(float));
  const uint32_t KKT_c_SIZE_BYTES = static_cast<uint32_t>((state_size * knot_points) * sizeof(float));
  const uint32_t KKT_FSTATES_SIZE_BYTES = static_cast<uint32_t>(fstates_size * sizeof(float));
  const uint32_t KKT_FCONTROL_SIZE_BYTES = static_cast<uint32_t>(fcontrol_size * sizeof(float));

  // const uint32_t DZ_SIZE_BYTES = static_cast<uint32_t>((states_s_controls * knot_points - control_size) * sizeof(float));

  float Q_R[((states_sq + controls_sq) * knot_points - controls_sq)];
  float q_r[((state_size + control_size) * knot_points - control_size)];
  float A_B[(states_sq + states_p_controls) * (knot_points - 1)];
  float d[(state_size * knot_points)];
  uint32_t soln_size = (state_size + state_size + control_size) * knot_points - control_size;
  float soln[soln_size];

  float my_soln[soln_size];

  // // Reading the LQR problem
  read_csv("../exmpls/lqr_prob8.csv", knot_points, state_size, control_size, Q_R, q_r, A_B, d);

  // Creating Factorization
  float F_lambda[fstates_size];
  float F_state[fstates_size];
  for (uint32_t n = 0; n < fstates_size; n++)
  {
    F_lambda[n] = 0;
    F_state[n] = 0;
  }

  float F_input[fcontrol_size];
  for (uint32_t n = 0; n < fcontrol_size; n++)
  {
    F_input[n] = 0;
  }

  // Allocate memory on the GPU for x0,Q_R,q_r, A_B, d,

  float *d_Q_R, *d_q_r, *d_A_B, *d_d,
      *d_F_lambda, *d_F_state, *d_F_input;

  gpuErrchk(hipMalloc((void **)&d_Q_R, KKT_G_DENSE_SIZE_BYTES));
  gpuErrchk(hipMalloc((void **)&d_q_r, KKT_g_SIZE_BYTES));
  gpuErrchk((hipMalloc((void **)&d_A_B, KKT_C_DENSE_SIZE_BYTES)));
  gpuErrchk(hipMalloc((void **)&d_d, KKT_c_SIZE_BYTES));
  gpuErrchk(hipMalloc((void **)&d_F_lambda, KKT_FSTATES_SIZE_BYTES));
  gpuErrchk(hipMalloc((void **)&d_F_state, KKT_FSTATES_SIZE_BYTES));
  gpuErrchk(hipMalloc((void **)&d_F_input, fcontrol_size * sizeof(float)));
  gpuErrchk(hipPeekAtLastError());

  // Copy the matrices from the host to the GPU memory
  // hipMemcpy(d_x0, x0, 6 * sizeof(float), hipMemcpyHostToDevice);
  gpuErrchk(hipMemcpy(d_Q_R, Q_R, KKT_G_DENSE_SIZE_BYTES, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_q_r, q_r, KKT_g_SIZE_BYTES, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_A_B, A_B, KKT_C_DENSE_SIZE_BYTES, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_d, d, KKT_c_SIZE_BYTES, hipMemcpyHostToDevice));

  gpuErrchk(hipMemcpy(d_F_lambda, F_lambda, KKT_FSTATES_SIZE_BYTES, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_F_state, F_state, KKT_FSTATES_SIZE_BYTES, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_F_input, F_input, KKT_FCONTROL_SIZE_BYTES, hipMemcpyHostToDevice));

  // Launch CUDA kernel with block and grid dimensions
  // find a way to automate number of threads and blocks
  std::uint32_t blockSize = 64;
  std::uint32_t gridSize = 8;

  //put it into a function
  uint32_t bchol_shared_mem_size = KKT_C_DENSE_SIZE_BYTES + KKT_G_DENSE_SIZE_BYTES + KKT_c_SIZE_BYTES + KKT_g_SIZE_BYTES +
                                   KKT_FCONTROL_SIZE_BYTES + KKT_FSTATES_SIZE_BYTES + KKT_FSTATES_SIZE_BYTES + (knot_points * 3 * sizeof(int))+500;

  std::cout << "shared_mem: " << bchol_shared_mem_size << std::endl;

  const void *bchol_kernelFunc = reinterpret_cast<const void *>(solve_BCHOL<float>);
  void *bchol_kernelArgs[] = {// prepare the kernel arguments
                              &knot_points,
                              &control_size,
                              &state_size,
                              &d_Q_R,
                              &d_q_r,
                              &d_A_B,
                              &d_d,
                              &d_F_lambda,
                              &d_F_state,
                              &d_F_input};
  // Prepare for timing
  hipEvent_t start, stop;
  float time;

  std::cout << "Launching blocks " << gridSize << " launching threads" << blockSize << "shared memory" << bchol_shared_mem_size << std::endl;
  if (DEBUG)
  {
    // Get device properties
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0); // Assuming you're using device 0

    std::cout << "Max threads per block: " << prop.maxThreadsPerBlock << std::endl;
    std::cout << "Max blocks per multiprocessor: " << prop.maxBlocksPerMultiProcessor << std::endl;
    std::cout << "Max shared memory per block: " << prop.sharedMemPerBlock << std::endl;
    std::cout << "Multiprocessor count: " << prop.multiProcessorCount << std::endl;

    // Calculate max blocks for cooperative launch
    int numBlocksPerSm;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, bchol_kernelFunc, blockSize, bchol_shared_mem_size);
    int maxBlocks = numBlocksPerSm * prop.multiProcessorCount;
    std::cout << "Max blocks for cooperative kernel launch: " << maxBlocks << std::endl;
  }
  gpuErrchk(hipEventCreate(&start));
  gpuErrchk(hipEventCreate(&stop));
  gpuErrchk(hipEventRecord(start, 0));
  gpuErrchk(hipDeviceSynchronize());
  gpuErrchk(hipLaunchCooperativeKernel(reinterpret_cast<const void*>(bchol_kernelFunc), gridSize, blockSize, bchol_kernelArgs, bchol_shared_mem_size));
  gpuErrchk(hipDeviceSynchronize());
  gpuErrchk(hipPeekAtLastError());

  printf("done with cuda!\n");

  // Copy back to the host
  gpuErrchk(hipMemcpy(q_r, d_q_r, KKT_g_SIZE_BYTES, hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(d, d_d, KKT_c_SIZE_BYTES, hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(Q_R, d_Q_R, KKT_G_DENSE_SIZE_BYTES, hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(A_B, d_A_B, KKT_C_DENSE_SIZE_BYTES, hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(F_lambda, d_F_lambda, KKT_FSTATES_SIZE_BYTES, hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(F_state, d_F_state, KKT_FSTATES_SIZE_BYTES, hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(F_input, d_F_input, KKT_FCONTROL_SIZE_BYTES, hipMemcpyDeviceToHost));

  gpuErrchk(hipEventRecord(stop, 0));
  gpuErrchk(hipEventSynchronize(stop));
  gpuErrchk(hipEventElapsedTime(&time, start, stop));
  printf("\nSolve Time:  %3.1f ms \n", time);

  for (uint32_t timestep = 0; timestep < knot_points; ++timestep)
  {
    for (uint32_t i = 0; i < state_size; ++i)
    {
      my_soln[timestep * (state_size + state_size + control_size) + i] = d[timestep * state_size + i];
    }
    for (uint32_t i = 0; i < states_s_controls; ++i)
    {
      my_soln[timestep * (state_size + state_size + control_size) + state_size + i] = q_r[timestep * (states_s_controls) + i];
    }
  }

  // if (checkEquality(my_soln, soln, soln_size))
  // {
  //   printf("PASSED!\n");
  // }
  // else
  // {
  //   printf("Not Passed");
  //   printf("my_soln\n");
  //   printMatrix(my_soln, (state_size + state_size + control_size) * 2, 1);
  //   printf("Soln\n");
  //   printMatrix(soln, (state_size + state_size + control_size) * 2, 1);
  // }

  std::cout << "size " << soln_size << std::endl;
  printMatrix(my_soln, soln_size, 1);

  // Free allocated GPU memory
  gpuErrchk(hipFree(d_Q_R));
  gpuErrchk(hipFree(d_q_r));
  gpuErrchk(hipFree(d_A_B));
  gpuErrchk(hipFree(d_d));
  gpuErrchk(hipFree(d_F_lambda));
  gpuErrchk(hipFree(d_F_state));
  gpuErrchk(hipFree(d_F_input));
}